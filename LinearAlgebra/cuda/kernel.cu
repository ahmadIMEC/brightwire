#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#define BLOCKSIZE 32
#define BLOCKSIZE2 BLOCKSIZE*BLOCKSIZE

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

extern "C"
{
	/*const float TOO_SMALL = -1.0E20f;
	const float TOO_BIG = 1.0E20f;

	__global__ float _Constrain(float d)
	{
		if (isnan(d))
			return 0;
		else if (isinf(d))
			return TOO_BIG;
		else if (d < TOO_SMALL)
			return TOO_SMALL;
		else if (d > TOO_BIG)
			return TOO_BIG;
		return d;
	}*/

	__global__ void PointwiseMultiply(float* a, float* b, int size)
	{
		int tidX = threadIdx.x;
		int index = blockDim.x * blockIdx.x + tidX;

		// read blockA into shared memory
		__shared__ float blockA[BLOCKSIZE2];
		if (index < size)
			blockA[tidX] = a[index];
		__syncthreads();

		if (index < size)
            b[index] *= blockA[tidX];
	}

	__global__ void PointwiseDivide(float* a, float* b, int size)
	{
		int tidX = threadIdx.x;
		int index = blockDim.x * blockIdx.x + tidX;

		// read blockA into shared memory
		__shared__ float blockA[BLOCKSIZE2];
		if (index < size)
			blockA[tidX] = a[index];
		__syncthreads();

		if (index < size)
            b[index] = blockA[tidX] / b[index];
	}

	__global__ void Sqrt(float* a, float* b, int size, float valueAdjustment)
	{
		int tidX = threadIdx.x;
		int index = blockDim.x * blockIdx.x + tidX;

		// read blockA into shared memory
		__shared__ float blockA[BLOCKSIZE2];
		if (index < size)
			blockA[tidX] = a[index];
		__syncthreads();

		if (index < size) {
            b[index] = sqrt(blockA[tidX] + valueAdjustment);
		}
	}

	__global__ void AddInPlace(float* a, float* b, int size, float coefficient1, float coefficient2)
	{
		int tidX = threadIdx.x;
		int index = blockDim.x * blockIdx.x + tidX;

		// read blockB into shared memory
		__shared__ float blockB[BLOCKSIZE2];
		if (index < size)
			blockB[tidX] = b[index];
		__syncthreads();

		if (index < size)
            a[index] = (a[index] * coefficient1) + (blockB[tidX] * coefficient2);
	}

	__global__ void SubtractInPlace(float* a, float* b, int size, float coefficient1, float coefficient2)
	{
		int tidX = threadIdx.x;
		int index = blockDim.x * blockIdx.x + tidX;

		// read blockB into shared memory
		__shared__ float blockB[BLOCKSIZE2];
		if (index < size)
			blockB[tidX] = b[index];
		__syncthreads();

		if (index < size)
            a[index] = (a[index] * coefficient1) - (blockB[tidX] * coefficient2);
	}

	__global__ void Transpose(float* a, float* b, int rows, int columns)
	{
		int i = BLOCKSIZE * blockIdx.x + threadIdx.x;
		int j = BLOCKSIZE * blockIdx.y + threadIdx.y;

		// read the data into shared memory
		__shared__ float block[BLOCKSIZE][BLOCKSIZE+1];
		if (i < rows && j < columns)
            block[threadIdx.y][threadIdx.x] = a[j * rows + i];
		__syncthreads();

		// write output
		i = blockIdx.y * BLOCKSIZE + threadIdx.x;
        j = blockIdx.x * BLOCKSIZE + threadIdx.y;
		if (i < columns && j < rows)
			b[j * columns + i] = block[threadIdx.x][threadIdx.y];
	}

	__global__ void InitData(float* a, int size, float value)
	{
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		if (index < size)
            a[index] = value;
	}

	__global__ void AddToEachRow(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns)
            a[j * rows + i] += b[j];
	}

	__global__ void AddToEachColumn(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns)
			a[j * rows + i] += b[i];
	}

	__global__ void TanH(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
			b[index] = tanh(a[index]);
		}
	}

	__global__ void TanHDerivative(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
			b[index] = 1.0f - pow(tanh(a[index]), 2);
		}
	}

	__global__ void Sigmoid(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
            b[index] = 1.0f / (1.0f + exp(-1.0f * a[index]));
		}
	}

	__global__ void SigmoidDerivative(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
            float sigmoid = 1.0f / (1.0f + exp(-1.0f * a[index]));
            b[index] = sigmoid * (1.0f - sigmoid);
        }
	}

	__global__ void RELU(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			b[index] = (val <= 0) ? 0 : val;
		}
	}

	__global__ void RELUDerivative(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			b[index] = (val <= 0) ? 0 : 1;
		}
	}

	__global__ void LeakyRELU(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			b[index] = (val <= 0) ? 0.01f*val : val;
		}
	}

	__global__ void LeakyRELUDerivative(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int index = j * rows + i;
            float val = a[index];
            b[index] = (val <= 0) ? 0.01f : 1;
        }
	}

	__global__ void SumRows(float* a, float* b, int rows, int columns)
	{
		// TODO: synchronised read?

		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < rows) {
            float temp = 0;
            for (int i = 0; i < columns; i++)
                temp += a[i * rows + row];
            b[row] = temp;
        }
	}

	__global__ void SumColumns(float* a, float* b, int rows, int columns)
	{
		// TODO: synchronised read?

		int column = blockIdx.x * blockDim.x + threadIdx.x;
		if (column < columns) {
            float temp = 0;
            for (int i = 0; i < rows; i++)
                temp += a[column * rows + i];
            b[column] = temp;
        }
	}

	__global__ void MemClear(float* data, int count, int srcOffset, int srcIncrement)
	{
		int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < count) {
            int arrayIndex = srcOffset + (index * srcIncrement);
            data[arrayIndex] = 0.0f;
        }
	}

	__global__ void MemCopy(float* a, float* b, int count, int srcOffset, int srcIncrement)
	{
		int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < count) {
            int arrayIndex = srcOffset + (index * srcIncrement);
            b[index] = a[arrayIndex];
        }
	}

	__global__ void SparseLoad(float* data, int* offset, int* length, int* destinationIndex, float* src, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		if (i < rows && j < columns) {
			int len = length[j];
			if(i < len) {
				int off = offset[j];
				int index = j * rows + destinationIndex[off + i];
				data[index] = src[off + i];
			}
		}
	}

	__global__ void FindMinAndMax(float* data, int count, float* minBlock, float* maxBlock)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		block[tidX] = (index < count) ? data[index] : 0;
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float min = FLT_MAX, max = FLT_MIN;
			int maxIndex = BLOCKSIZE2;
			if (count - index < BLOCKSIZE2)
				maxIndex = count - index;
			for (int i = 0; i < maxIndex; i++) {
				float val = block[i];
				if (val > max)
					max = val;
				if (val < min)
					min = val;
			}
			minBlock[blockX] = min;
			maxBlock[blockX] = max;
		}
	}

	__global__ void FindSum(float* data, int count, float* sum)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += block[i];
			}
			sum[blockX] = total;
		}
	}

	__global__ void FindStdDev(float* data, int count, float mean, float* stdDev)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += pow(block[i] - mean, 2);
			}
			stdDev[blockX] = total;
		}
	}

	__global__ void Constrain(float* data, int count, float min, float max)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			float val = data[i];
			if (val < min)
				data[i] = min;
			if (val > max)
				data[i] = max;
		}
	}

	__global__ void Pow(float* a, float* b, int count, float power)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			float val = a[i];
			b[i] = pow(val, power);
		}
	}

	__global__ void Diagonal(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < rows && i < columns) {
			b[i] = a[i * rows + i];
		}
	}

	__global__ void L1Regularisation(float* a, int count, float coefficient)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			float val = a[i];
			a[i] = val - ((val > 0 ? 1 : val < 0 ? -1 : 0) * coefficient);
		}
	}

	__global__ void PointwiseDivideRows(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			a[index] = val / b[i];
		}
	}

	__global__ void PointwiseDivideColumns(float* a, float* b, int rows, int columns)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			a[index] = val / b[j];
		}
	}

	__global__ void SplitRows(float* a, float* b, float* c, int rows, int columns, int position)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			int index = j * rows + i;
			float val = a[index];
			if(j >= position) {
				int diff = j - position;
				c[diff * rows + i] = val;
			}else
				b[index] = val;
		}
	}

	__global__ void SplitColumns(float* a, float* b, float* c, int rows, int columns, int position)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val = a[j * rows + i];
			if(i >= position) {
				int diff = i - position;
				c[j * (rows-position) + diff] = val;
			}else
				b[j * position + i] = val;
		}
	}

	__global__ void ConcatColumns(float* a, float* b, float* c, int rows, int columns, int topRowCount, int bottomRowCount)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val;
			if(i >= topRowCount)
				val = b[j * bottomRowCount + i - topRowCount];
			else
				val = a[j * topRowCount + i];
			c[j * rows + i] = val;
		}
	}

	__global__ void ConcatRows(float* a, float* b, float* c, int rows, int columns, int leftColumnCount)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;

		if (i < rows && j < columns) {
			float val;
			if(j >= leftColumnCount)
				val = b[(j-leftColumnCount) * rows + i];
			else
				val = a[j * rows + i];
			c[j * rows + i] = val;
		}
	}

	__global__ void EuclideanDistance(float* a, float* b, float* c, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			c[i] = pow(a[i] - b[i], 2);
		}
	}

	__global__ void ManhattanDistance(float* a, float* b, float* c, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			c[i] = abs(a[i] - b[i]);
		}
	}

	__global__ void Abs(float* a, float* b, int count)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			b[i] = abs(a[i]);
		}
	}

	__global__ void Normalise(float* a, int count, float min, float range)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < count) {
			a[i] = (a[i] - min) / range;
		}
	}
}